#include "hip/hip_runtime.h"
/*
 * $Id$
 *
 * Author: <burkhard.heisen@xfel.eu>
 *
 * Created on May 30, 2011, 7:43 PM
 *
 * Copyright (C) European XFEL GmbH Hamburg. All rights reserved.
 */

/*
 * $Id$
 *
 * Author: <burkhard.heisen@xfel.eu>
 *
 * Created on May 25, 2011, 8:57 PM
 *
 * Copyright (C) European XFEL GmbH Hamburg. All rights reserved.
 */

#ifndef EXFEL_XIP_GPUIMAGE_CU
#define	EXFEL_XIP_GPUIMAGE_CU

#include <exfel/util/Types.hh>
#include <hip/hip_runtime_api.h>

namespace ut = exfel::util;

// Multiprocessors
#define BLOCK_SIZE 16

template<typename T>
__global__ void kernel_fill(T* img, int size, T value) {
  const int gridSize = blockDim.x * gridDim.x;
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  while (idx < size) {
    img[idx] = value;
    idx += gridSize;
  };
}

extern "C"
void cudaFill(ut::Types::Type type, void* img, int size, void* value) {
  switch (type) {
    case ut::Types::INT16:
      kernel_fill << <64, 128 >> > ((short*)img, size, *((short*)value));
      break;
    case ut::Types::INT32:
      kernel_fill << <64, 128 >> > ((int*)img, size, *((int*)value));
      break;
    case ut::Types::FLOAT:
      kernel_fill << <64, 128 >> > ((float*)img, size, *((float*)value));
      break;
  }
}



#endif

